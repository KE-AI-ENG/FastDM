#include "hip/hip_runtime.h"
// Adapted from
// https://github.com/vllm-project/vllm/blob/main/csrc/quantization/cutlass_w8a8/c3x/cutlass_gemm_caller.cuh
// https://github.com/vllm-project/vllm/blob/main/csrc/quantization/cutlass_w8a8/c3x/scaled_mm_sm90_int8_dispatch.cuh

#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/util/packed_stride.hpp"

#include "scaled_mm_c3x.cuh"

#include "cutlass_extensions/epilogue/scaled_mm_epilogues_c3x.hpp"

/**
 * This file defines Gemm kernel configurations for SM90 (int8) based on the
 * Gemm shape.
 */

namespace fastdm {

static inline cute::Shape<int, int, int, int> get_problem_shape(
    torch::Tensor const& a, torch::Tensor const& b) {
  int32_t m = a.size(0), n = b.size(1), k = a.size(1);
  return {m, n, k, 1};
}

template <typename GemmKernel>
void cutlass_gemm_caller(
    torch::Device device, cute::Shape<int, int, int, int> prob_shape,
    typename GemmKernel::MainloopArguments mainloop_args,
    typename GemmKernel::EpilogueArguments epilogue_args,
    typename GemmKernel::TileSchedulerArguments scheduler = {}) {
  cutlass::KernelHardwareInfo hw_info;
  typename GemmKernel::Arguments args{cutlass::gemm::GemmUniversalMode::kGemm,
                                      prob_shape,
                                      mainloop_args,
                                      epilogue_args,
                                      hw_info,
                                      scheduler};

  // Launch the CUTLASS GEMM kernel.
  using GemmOp = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  GemmOp gemm_op;
  CUTLASS_CHECK(gemm_op.can_implement(args));

  size_t workspace_size = gemm_op.get_workspace_size(args);
  auto const workspace_options =
      torch::TensorOptions().dtype(torch::kUInt8).device(device);
  auto workspace = torch::empty(workspace_size, workspace_options);

  auto stream = at::cuda::getCurrentCUDAStream(device.index());

  cutlass::Status status = gemm_op.run(args, workspace.data_ptr(), stream);
  CUTLASS_CHECK(status);
}

template <typename Gemm, typename... EpilogueArgs>
void cutlass_gemm_caller(torch::Tensor& out, torch::Tensor const& a,
                         torch::Tensor const& b,
                         EpilogueArgs&&... epilogue_params) {
  using ElementAB = typename Gemm::ElementAB;
  using ElementC = typename Gemm::ElementC;
  using ElementD = typename Gemm::ElementD;
  using GemmKernel = typename Gemm::GemmKernel;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = StrideC;
  using StrideAux = StrideC;

  typename GemmKernel::ProblemShape prob_shape = get_problem_shape(a, b);
  auto [M, N, K, L] = prob_shape;

  StrideA a_stride =
      cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));
  StrideB b_stride =
      cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, L));
  StrideC c_stride =
      cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, L));
  StrideD d_stride =
      cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(M, N, L));
  StrideAux aux_stride = d_stride;

  auto a_ptr = static_cast<ElementAB*>(a.data_ptr());
  auto b_ptr = static_cast<ElementAB*>(b.data_ptr());
  typename GemmKernel::MainloopArguments mainloop_args{a_ptr, a_stride, b_ptr,
                                                       b_stride};

  auto c_ptr = static_cast<ElementD*>(out.data_ptr());
  // auto d_ptr = static_cast<ElementC*>(out.data_ptr());
  typename GemmKernel::EpilogueArguments epilogue_args{
      Gemm::Epilogue::prepare_args(
          std::forward<EpilogueArgs>(epilogue_params)...),
      c_ptr, c_stride, c_ptr, d_stride};

  cutlass_gemm_caller<GemmKernel>(a.device(), prob_shape, mainloop_args,
                                  epilogue_args);
}

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue>
struct sm90_int8_config_default {
  // For M > 128 and any N
  static_assert(std::is_same<InType, int8_t>());
  using KernelSchedule =
      typename cutlass::gemm::KernelTmaWarpSpecializedPingpong;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_128, _128, _128>;
  using ClusterShape = Shape<_2, _1, _1>;
  using Cutlass3xGemm =
      cutlass_3x_gemm<InType, OutType, Epilogue, TileShape, ClusterShape,
                      KernelSchedule, EpilogueSchedule>;
};

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue>
struct sm90_int8_config_M128 {
  // For M in (64, 128] and any N
  static_assert(std::is_same<InType, int8_t>());
  using KernelSchedule =
      typename cutlass::gemm::KernelTmaWarpSpecializedPingpong;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_64, _128, _128>;
  using ClusterShape = Shape<_2, _1, _1>;
  using Cutlass3xGemm =
      cutlass_3x_gemm<InType, OutType, Epilogue, TileShape, ClusterShape,
                      KernelSchedule, EpilogueSchedule>;
};

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue>
struct sm90_int8_config_M64 {
  // For M in (32, 64] and any N
  static_assert(std::is_same<InType, int8_t>());
  using KernelSchedule = typename cutlass::gemm::KernelTmaWarpSpecialized;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_64, _64, _256>;
  using ClusterShape = Shape<_1, _1, _1>;
  using Cutlass3xGemm =
      cutlass_3x_gemm<InType, OutType, Epilogue, TileShape, ClusterShape,
                      KernelSchedule, EpilogueSchedule>;
};

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue>
struct sm90_int8_config_M32_NBig {
  // For M in [1, 32] and N >= 8192
  static_assert(std::is_same<InType, int8_t>());
  using KernelSchedule = typename cutlass::gemm::KernelTmaWarpSpecialized;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_64, _128, _256>;
  using ClusterShape = Shape<_1, _4, _1>;
  using Cutlass3xGemm =
      cutlass_3x_gemm<InType, OutType, Epilogue, TileShape, ClusterShape,
                      KernelSchedule, EpilogueSchedule>;
};

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue>
struct sm90_int8_config_M32_NSmall {
  // For M in [1, 32] and N < 8192
  static_assert(std::is_same<InType, int8_t>());
  using KernelSchedule = typename cutlass::gemm::KernelTmaWarpSpecialized;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_64, _64, _256>;
  using ClusterShape = Shape<_1, _8, _1>;
  using Cutlass3xGemm =
      cutlass_3x_gemm<InType, OutType, Epilogue, TileShape, ClusterShape,
                      KernelSchedule, EpilogueSchedule>;
};

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue,
          typename... EpilogueArgs>
inline void cutlass_gemm_sm90_int8_dispatch(torch::Tensor& out,
                                            torch::Tensor const& a,
                                            torch::Tensor const& b,
                                            EpilogueArgs&&... args) {
  static_assert(std::is_same<InType, int8_t>());
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);

  using Cutlass3xGemmDefault =
      typename sm90_int8_config_default<InType, OutType,
                                        Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM128 =
      typename sm90_int8_config_M128<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM64 =
      typename sm90_int8_config_M64<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM32NBig =
      typename sm90_int8_config_M32_NBig<InType, OutType,
                                         Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM32NSmall =
      typename sm90_int8_config_M32_NSmall<InType, OutType,
                                           Epilogue>::Cutlass3xGemm;

  uint32_t const n = out.size(1);
  bool const is_small_n = n < 8192;

  uint32_t const m = a.size(0);
  uint32_t const mp2 =
      std::max(static_cast<uint32_t>(32), next_pow_2(m));  // next power of 2

  if (mp2 <= 32) {
    // m in [1, 32]
    if (is_small_n) {
      return cutlass_gemm_caller<Cutlass3xGemmM32NSmall>(
          out, a, b, std::forward<EpilogueArgs>(args)...);
    } else {
      return cutlass_gemm_caller<Cutlass3xGemmM32NBig>(
          out, a, b, std::forward<EpilogueArgs>(args)...);
    }
  } else if (mp2 <= 64) {
    // m in (32, 64]
    return cutlass_gemm_caller<Cutlass3xGemmM64>(
        out, a, b, std::forward<EpilogueArgs>(args)...);
  } else if (mp2 <= 128) {
    // m in (64, 128]
    return cutlass_gemm_caller<Cutlass3xGemmM128>(
        out, a, b, std::forward<EpilogueArgs>(args)...);
  } else {
    // m in (128, inf)
    return cutlass_gemm_caller<Cutlass3xGemmDefault>(
        out, a, b, std::forward<EpilogueArgs>(args)...);
  }
}

template <template <typename, typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_scaled_mm_sm90_int8_epilogue(torch::Tensor& out,
                                          torch::Tensor const& a,
                                          torch::Tensor const& b,
                                          EpilogueArgs&&... epilogue_args) {
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);

  if (out.dtype() == torch::kBFloat16) {
    return cutlass_gemm_sm90_int8_dispatch<int8_t, cutlass::bfloat16_t,
                                           Epilogue>(
        out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return cutlass_gemm_sm90_int8_dispatch<int8_t, cutlass::half_t, Epilogue>(
        out, a, b, std::forward<EpilogueArgs>(epilogue_args)...);
  }
}

}  // namespace fastdm

using namespace fastdm;

void int8_scaled_mm_sm90(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& b,
                                     torch::Tensor const& a_scales,
                                     torch::Tensor const& b_scales,
                                     torch::Tensor const& azp_adj,
                                     torch::Tensor const& azp,
                                     std::optional<torch::Tensor> const& bias) {
    return cutlass_scaled_mm_sm90_int8_epilogue<
        c3x::ScaledEpilogueBiasAzpToken>(out, a, b, a_scales, b_scales, azp_adj,
                                         azp, bias);
}